#include "hip/hip_runtime.h"
#include "lbm.h"

__global__ void kernel_func() {
    
}

void lbm_d2q9(){
    float cabinet_size = 1.0; // 正方キャビネットの辺の長さ[m]

    int lattice_num = 100; // 1辺の格子点の数

    float delta_r = cabinet_size / (float)lattice_num; // 離散化間隔[m]
    float delta_t = 0.1;                               // 離散化間隔[sec]

    dim3 block(9, 1, 1);
    dim3 grid()

}

