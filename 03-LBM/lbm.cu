#include "hip/hip_runtime.h"
#include "lbm.h"
#include "calc.h"

__global__ void kernel_func() {
    static f[9] = {
        
    };
}

// 局所平衡分布関数を計算
float* calc_f_eq(float* v, float rho){
    e[9][2] = {{0,0},{1,0},{0,1},{-1,0},{0,-1},{1,1},{-1,1},{-1,-1},{1,-1}};
    
    float f_eq[9];
    f_eq[0] = 4.0/ 9.0 * rho * (1.0 - 3.0/2.0*(dot(2,v,v)) + 3.0*dot(2,e[0],v) + 9.0/2.0*(dot(2,e[0],v))*(dot(2,e[0],v)) );
    f_eq[1] = 1.0/ 9.0 * rho * (1.0 - 3.0/2.0*(dot(2,v,v)) + 3.0*dot(2,e[1],v) + 9.0/2.0*(dot(2,e[1],v))*(dot(2,e[1],v)) );
    f_eq[2] = 1.0/ 9.0 * rho * (1.0 - 3.0/2.0*(dot(2,v,v)) + 3.0*dot(2,e[2],v) + 9.0/2.0*(dot(2,e[2],v))*(dot(2,e[2],v)) );
    f_eq[3] = 1.0/ 9.0 * rho * (1.0 - 3.0/2.0*(dot(2,v,v)) + 3.0*dot(2,e[3],v) + 9.0/2.0*(dot(2,e[3],v))*(dot(2,e[3],v)) );
    f_eq[4] = 1.0/ 9.0 * rho * (1.0 - 3.0/2.0*(dot(2,v,v)) + 3.0*dot(2,e[4],v) + 9.0/2.0*(dot(2,e[4],v))*(dot(2,e[4],v)) );
    f_eq[5] = 1.0/36.0 * rho * (1.0 - 3.0/2.0*(dot(2,v,v)) + 3.0*dot(2,e[5],v) + 9.0/2.0*(dot(2,e[5],v))*(dot(2,e[5],v)) );
    f_eq[6] = 1.0/36.0 * rho * (1.0 - 3.0/2.0*(dot(2,v,v)) + 3.0*dot(2,e[6],v) + 9.0/2.0*(dot(2,e[6],v))*(dot(2,e[6],v)) );
    f_eq[7] = 1.0/36.0 * rho * (1.0 - 3.0/2.0*(dot(2,v,v)) + 3.0*dot(2,e[7],v) + 9.0/2.0*(dot(2,e[7],v))*(dot(2,e[7],v)) );
    f_eq[7] = 1.0/36.0 * rho * (1.0 - 3.0/2.0*(dot(2,v,v)) + 3.0*dot(2,e[8],v) + 9.0/2.0*(dot(2,e[8],v))*(dot(2,e[8],v)) );

    return f_eq;
}

// 密度rhoの計算
float calc_rho(float* f){
    float rho = 0.0;
    for(int i=0; i<9; i++){
        rho += f[i];
    }
    return rho;
}

// 流速vの計算
float* calc_v(float delta_r, float delta_time, float rho, float* f){
    float v[2] = {0.0, 0.0};
    float c = delta_r / delta_time;
    e[9][2] = {{0,0},{1,0},{0,1},{-1,0},{0,-1},{1,1},{-1,1},{-1,-1},{1,-1}};

    for(int i=0; i<9; i++){
        v[0] += c / rho * f[i] * e[i][0];
        v[1] += c / rho * f[i] * e[i][1];
    }

    return v;
}

void streaming(){

}

void apply_boundary_conditions(){

}

void collision(){

}

void lbm_d2q9(){
    float cabinet_size = 1.0; // 正方キャビネットの辺の長さ[m]

    int lattice_num = 1000; // 1辺の格子点の数

    float delta_r = cabinet_size / (float)(lattice_num - 1 ); // 離散化間隔[m]
    float delta_time = 0.1;                                      // 離散化間隔[sec]

    dim3 block(10, 1, 1);
    dim3 grid(lattice_num*lattice_num/block.x)

    streaming();
    apply_boundary_conditions();
    collision();

}

