#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hello.h"

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

void hello(){
    cuda_hello<<<2,4>>>();
    hipDeviceSynchronize();
}
