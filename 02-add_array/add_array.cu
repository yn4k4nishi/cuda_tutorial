#include "hip/hip_runtime.h"
#include <stdio.h>
#include "add_array.h"

__global__ void add_cuda(int *a, int *b, int *c){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] = a[i] + b[i];

    printf("%d\n",i);
}

void add_array(int n, int *a, int *b, int *c){
    int *d_a, *d_b, *d_c; // pointer for GPU

    // assign GPU memory
    hipMalloc(&d_a, n*sizeof(int));
    hipMalloc(&d_b, n*sizeof(int));
    hipMalloc(&d_c, n*sizeof(int));

    // memory copy from CPU to GPU
    hipMemcpy(d_a, a, n*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, n*sizeof(int), hipMemcpyHostToDevice);

    // call kernel function
    int block_size = 512;
    dim3 block(block_size,1,1);
    dim3 grid(n/block_size,1,1);

    add_cuda<<<grid, block>>>(d_a, d_b, d_c);
    hipDeviceSynchronize();

    // memory copy from GPU to CPU
    hipMemcpy(c, d_c, n*sizeof(int), hipMemcpyDeviceToHost);

    // free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
